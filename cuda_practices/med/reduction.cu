#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void parallel_reduction(const float* input, const float* output, int N) {
    // combine halved idx 
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x; 

    __shared__ unsigned long long data[blockDim.x];

    data[threadIdx.x] = (global_idx < N) ? input[global_idx] : 0; 
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        if (threadIdx.x < stride) {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    if (threadIdx.x == 0) {
        atomicAdd(&output, data[0]);
    }
}

// input, output are device pointers
void solve(const float* input, float* output, int N) {  
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    parallel_reduction<<<blocksPerGrid, threadsPerBlock>>>(input, output, N); 
    hipDeviceSynchronize();
}