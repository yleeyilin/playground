#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void relu_kernel(const float* input, float* output, int N) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_idx < N) {
        output[global_idx] =  input[global_idx] < 0.0 ? 0.0 : input[global_idx]; 
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}
