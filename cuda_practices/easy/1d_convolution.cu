#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void convolution_1d_kernel(const float* input, const float* kernel, float* output,
                                      int input_size, int kernel_size) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_idx <= input_size - kernel_size) {
        int res = 0;
        for (int i = 0; i < kernel_size; i++) {
            res = res + input[global_idx + i] * kernel[i]; 
        }
        output[global_idx] = res; 
    }
}

// input, kernel, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, const float* kernel, float* output, int input_size, int kernel_size) {
    int output_size = input_size - kernel_size + 1;
    int threadsPerBlock = 256;
    int blocksPerGrid = (output_size + threadsPerBlock - 1) / threadsPerBlock;

    convolution_1d_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, kernel, output, input_size, kernel_size);
    hipDeviceSynchronize();
}