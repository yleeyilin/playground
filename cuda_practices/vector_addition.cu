#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_idx < N) {
        C[global_idx] =  A[global_idx] + B[global_idx]; 
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();
}
